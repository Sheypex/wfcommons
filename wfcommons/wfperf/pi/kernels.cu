#include "hip/hip_runtime.h"
#include "kernels.cuh"
#include <stdio.h>

__global__ void setup_kernel(hiprandState *state)
{
	int index = threadIdx.x + blockDim.x * blockIdx.x;
    hiprand_init(123456789, index, 0, &state[index]);
}

__global__ void monte_carlo_kernel(hiprandState *state, int *d_count, int m)
{
	unsigned int index_x = threadIdx.x + blockDim.x*blockIdx.x;
	// unsigned int index_y = threadIdx.y + blockDim.y*blockIdx.y;
	
	__shared__ int inside_circle[NB_THREADS_X]; //Data shared per block
	inside_circle[threadIdx.x] = 0;
	
	unsigned int temp = 0;
	while(temp < m){
		float x = hiprand_uniform(&state[index_x]);
		float y = hiprand_uniform(&state[index_x]);
		if (x*x + y*y <= 1.0f) {
			inside_circle[threadIdx.x]++;
		}
		temp++;
	}

	#ifdef COMPUTE_PI
	// We actually compute Pi
	// Reduction on threads for each block
	for(int i = 1; i < blockDim.x; i *= 2) {
		if (threadIdx.x % (i*2) == 0) {
			inside_circle[threadIdx.x] += inside_circle[threadIdx.x + i];
		}
		__syncthreads();
	}

	// update to our global variable count for each block (done by thread 0)
	if(threadIdx.x == 0) {
		atomicAdd(d_count, inside_circle[0]);
	}
	#endif
}



