#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <chrono>
#include <thread>
#include <hiprand.h>
#include <ctime>
#include <cmath>
#include "kernels.cuh"

double PI = 3.1415926535897932384626433832795028841971693993751058209749445923;

int main(int argc, char** argv)
{
    if (argc != 2) {
        std::cerr << "Usage: " << argv[0] << " <work (# 1M samples)>\n";
        exit(1);
    }

    unsigned int n = NB_BLOCKS_X*NB_THREADS_X*NB_BLOCKS_X*NB_THREADS_X;
    unsigned int m;
    unsigned int work;
    #ifdef COMPUTE_PI
    int *h_count;
    #endif
    int *d_count;
    hiprandState *d_state;

    int numdevices;
    int numprocs;
    int grid_dim_x;
    int grid_dim_y;
    int grid_dim_z;
    int block_dim_x;
    int block_dim_y;
    int block_dim_z;
    int max_thread_block;
    int current_id;

    //checking the user input for work
    try {
        work = std::stoi(argv[1]);
    } catch (std::invalid_argument &e) {
        std::cerr << "Invalid argument: " << e.what() << "\n";
        exit(1);
    }
    
    hipGetDeviceCount(&numdevices);
    hipGetDevice(&current_id);

    hipDeviceGetAttribute(&numprocs, hipDeviceAttributeMultiprocessorCount, current_id);
    hipDeviceGetAttribute(&max_thread_block, hipDeviceAttributeMaxThreadsPerBlock, current_id);
    hipDeviceGetAttribute(&grid_dim_x, hipDeviceAttributeMaxBlockDimX, current_id);
    hipDeviceGetAttribute(&grid_dim_y, hipDeviceAttributeMaxBlockDimY, current_id);
    hipDeviceGetAttribute(&grid_dim_z, hipDeviceAttributeMaxBlockDimZ, current_id);
    hipDeviceGetAttribute(&block_dim_x, hipDeviceAttributeMaxGridDimX, current_id);
    hipDeviceGetAttribute(&block_dim_y, hipDeviceAttributeMaxGridDimY, current_id);
    hipDeviceGetAttribute(&block_dim_z, hipDeviceAttributeMaxGridDimZ, current_id);

    std::cout << "Number of GPUs: " << numdevices << std::endl;
    std::cout << "Current GPU ID: " << current_id << std::endl;
    std::cout << "   Number of multiprocessors: " << numprocs << std::endl;
    std::cout << "   Grid max size: (" << grid_dim_x << "," << grid_dim_y << "," << grid_dim_z << ")"  << std::endl;
    std::cout << "   Block max size: (" << block_dim_x << "," << block_dim_y << "," << block_dim_z << ")"  << std::endl;
    std::cout << "   Max thread per blocks: " << max_thread_block << std::endl;

    // set up timing stuff
    float gpu_elapsed_time;
    hipEvent_t gpu_start, gpu_stop;
    hipEventCreate(&gpu_start);
    hipEventCreate(&gpu_stop);
    hipEventRecord(gpu_start, 0);

    //making into M samples
    m = 1000000*work;
    // allocate memory
    #ifdef COMPUTE_PI
    h_count = (int*)malloc(n*sizeof(int));
    #endif
    hipMalloc((void**)&d_count, NB_BLOCKS_X*sizeof(int));
    hipMalloc((void**)&d_state, n*sizeof(hiprandState));
    hipMemset(d_count, 0, sizeof(int));

    // set kernel
    dim3 gridSize(NB_BLOCKS_X,1,1);
    dim3 blockSize(NB_THREADS_X,1,1);

    setup_kernel<<< gridSize, blockSize>>>(d_state);
    // monte carlo kernel
    monte_carlo_kernel<<<gridSize, blockSize>>>(d_state, d_count, m);

    // // copy results back to the host
    #ifdef COMPUTE_PI
    hipMemcpy(h_count, d_count, NB_BLOCKS_X*sizeof(int), hipMemcpyDeviceToHost);
    #endif
    hipEventRecord(gpu_stop, 0);
    hipEventSynchronize(gpu_stop);
    hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
    hipEventDestroy(gpu_start);
    hipEventDestroy(gpu_stop);

    #ifdef COMPUTE_PI
    // display results and timings for gpu
    double pi = 0.0;
    for (int i = 0; i < NB_BLOCKS_X; i++) {
        pi += (double) h_count[i];
    }
    pi = 4.0*pi/(m*NB_BLOCKS_X*NB_THREADS_X);
    std::cout << "Approximate pi calculated on GPU is: " << pi << std::setprecision(6) << " (relative error: " << fabs((PI-pi)/PI) << ")" << std::endl;
    #endif
    
    std::cout << std::setprecision(6) << "GPU stress test is over and it took " << gpu_elapsed_time/1000.0 << " seconds" << std::endl;

    // delete memory
    #ifdef COMPUTE_PI
    free(h_count);
    #endif
    hipFree(d_count);
    hipFree(d_state);
}

